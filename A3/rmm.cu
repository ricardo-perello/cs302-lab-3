/*
============================================================================
Filename    : rmm.cu
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

/* CPU Baseline */
void rmm_cpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    for(int idx = 0; idx < M/2; idx++) {
        for(int jdx = 0; jdx < K/2; jdx++) {
            matC[idx*(K/2) + jdx] = 0;
            for(int aoff = 0; aoff < 2; aoff++) {
                for(int boff = 0; boff < 2; boff++) {
                    for(int kdx = 0; kdx < N; kdx++) {
                        matC[idx*(K/2) + jdx] += matA[(idx*2 + aoff)*N + kdx] * matB[kdx*K + jdx*2 + boff];
                    }
                }
            }
        }
    }
}

/* GPU Optimized Function */
void rmm_gpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    /* Cuda events for calculating elapsed time */
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);

    /* Preprocessing (if any) goes here */

    hipEventRecord(cpy_H2D_start);
    /* Copying array(s) from host to device goes here */
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    hipEventRecord(comp_start);
    /* Launching the GPU kernel to do the computation goes here */
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array(s) from device to host goes here */
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing (if any) goes here */

    /* Display timing statistics */
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "RMM operation takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;
}