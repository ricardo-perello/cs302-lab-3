/*
============================================================================
Filename    : rmm.cu
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

/* CPU Baseline */
void rmm_cpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    for(int idx = 0; idx < M/2; idx++) {
        for(int jdx = 0; jdx < K/2; jdx++) {
            matC[idx*(K/2) + jdx] = 0;
            for(int aoff = 0; aoff < 2; aoff++) {
                for(int boff = 0; boff < 2; boff++) {
                    for(int kdx = 0; kdx < N; kdx++) {
                        matC[idx*(K/2) + jdx] += matA[(idx*2 + aoff)*N + kdx] * matB[kdx*K + jdx*2 + boff];
                    }
                }
            }
        }
    }
}

#define TILE 16

__global__ void rmm_kernel(
    const int* __restrict__ A,
    const int* __restrict__ B,
          int* __restrict__ C,
    int M, int N, int K)
{
    // global-reduced-matrix coords
    int out_row = blockIdx.y * TILE + threadIdx.y;  // 0 .. M/2-1
    int out_col = blockIdx.x * TILE + threadIdx.x;  // 0 .. K/2-1

    // mark who's actually computing a valid C element
    bool active = (out_row < M/2 && out_col < K/2);

    // precompute the two A-row bases and two B-col offsets (even if inactive)
    int a0_base = (out_row*2    ) * N;
    int a1_base = (out_row*2 + 1) * N;
    int b0_off  = out_col*2;
    int b1_off  = b0_off + 1;

    int sum = 0;

    // shared-memory tiles: 2 rows × TILE columns, and TILE rows × 2 columns
    __shared__ int sA[2*TILE][TILE];
    __shared__ int sB[TILE][2*TILE];

    int numTiles = (N + TILE - 1) / TILE;
    for (int t = 0; t < numTiles; ++t) {
        int kBase = t * TILE;

        // --- load A (two rows) ---
        int col = kBase + threadIdx.x;
        if (col < N) {
            sA[threadIdx.y*2    ][threadIdx.x] = A[a0_base + col];
            sA[threadIdx.y*2 + 1][threadIdx.x] = A[a1_base + col];
        } else {
            sA[threadIdx.y*2    ][threadIdx.x] = 0;
            sA[threadIdx.y*2 + 1][threadIdx.x] = 0;
        }

        // --- load B (two cols) ---
        int brow = kBase + threadIdx.y;
        if (brow < N) {
            sB[threadIdx.y][threadIdx.x*2    ] = B[brow*K + b0_off];
            sB[threadIdx.y][threadIdx.x*2 + 1] = B[brow*K + b1_off];
        } else {
            sB[threadIdx.y][threadIdx.x*2    ] = 0;
            sB[threadIdx.y][threadIdx.x*2 + 1] = 0;
        }

        // *all* threads sync before we read from sA/sB
        __syncthreads();

        // accumulate only if this thread is active
        if (active) {
            int limit = min(TILE, N - kBase);
            for (int k = 0; k < limit; ++k) {
                int A0 = sA[threadIdx.y*2    ][k];
                int A1 = sA[threadIdx.y*2 + 1][k];
                int B0 = sB[k][threadIdx.x*2    ];
                int B1 = sB[k][threadIdx.x*2 + 1];
                sum += A0*B0 + A0*B1 + A1*B0 + A1*B1;
            }
        }

        // *all* threads sync before we overwrite sA/sB
        __syncthreads();
    }

    // finally write C if active
    if (active) {
        C[out_row * (K/2) + out_col] = sum;
    }
}

/* GPU Optimized Function */
void rmm_gpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    /* Cuda events for calculating elapsed time */
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);

    // Allocate device memory
    int *d_matA, *d_matB, *d_matC;
    hipMalloc(&d_matA, M * N * sizeof(int));
    hipMalloc(&d_matB, N * K * sizeof(int));
    hipMalloc(&d_matC, (M/2) * (K/2) * sizeof(int));

    hipEventRecord(cpy_H2D_start);
    // Copy input matrices to device
    hipMemcpy(d_matA, matA, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, N * K * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    // Calculate grid and block dimensions
    dim3 blockDim(TILE, TILE);  // TILExTILE threads per block
    dim3 gridDim((K/2 + TILE - 1) / TILE, 
                 (M/2 + TILE - 1) / TILE);

    hipEventRecord(comp_start);
    // Launch kernel
    rmm_kernel<<<gridDim, blockDim>>>(d_matA, d_matB, d_matC, M, N, K);
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    // Copy result back to host
    hipMemcpy(matC, d_matC, (M/2) * (K/2) * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    /* Display timing statistics */
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "RMM operation takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;
}