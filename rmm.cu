/*
============================================================================
Filename    : rmm.cu
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

/* CPU Baseline */
void rmm_cpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    for(int idx = 0; idx < M/2; idx++) {
        for(int jdx = 0; jdx < K/2; jdx++) {
            matC[idx*(K/2) + jdx] = 0;
            for(int aoff = 0; aoff < 2; aoff++) {
                for(int boff = 0; boff < 2; boff++) {
                    for(int kdx = 0; kdx < N; kdx++) {
                        matC[idx*(K/2) + jdx] += matA[(idx*2 + aoff)*N + kdx] * matB[kdx*K + jdx*2 + boff];
                    }
                }
            }
        }
    }
}

#define TILE 16

__global__ void rmm_kernel(
    const int* __restrict__ A,
    const int* __restrict__ B,
          int* __restrict__ C,
    int M, int N, int K)
{
    // 1) Compute which reduced-matrix element this thread handles
    int out_row = blockIdx.y * TILE + threadIdx.y;  // 0..M/2-1
    int out_col = blockIdx.x * TILE + threadIdx.x;  // 0..K/2-1

    if (out_row >= M/2 || out_col >= K/2) return;

    // 2) Precompute the two source-row bases in A and two source-col offsets in B
    int a0_base = (out_row*2    ) * N;
    int a1_base = (out_row*2 + 1) * N;
    int b0_off  = out_col*2;
    int b1_off  = b0_off + 1;

    int sum = 0;

    // 3) Allocate shared memory for a TILE×TILE slice of A×B,
    //    but folded so each thread loads exactly two A values and two B values
    __shared__ int sA[2*TILE][TILE];
    __shared__ int sB[TILE][2*TILE];

    // 4) Loop over N in TILE-sized chunks
    int numTiles = (N + TILE - 1) / TILE;
    for (int t = 0; t < numTiles; ++t) {
        int kBase = t * TILE;

        // 4a) Load A: two rows per out_row, one element each
        int col = kBase + threadIdx.x;
        if (col < N) {
            sA[threadIdx.y*2    ][threadIdx.x] = A[a0_base + col];
            sA[threadIdx.y*2 + 1][threadIdx.x] = A[a1_base + col];
        } else {
            sA[threadIdx.y*2    ][threadIdx.x] = 0;
            sA[threadIdx.y*2 + 1][threadIdx.x] = 0;
        }

        // 4b) Load B: two cols per out_col, one element each
        int brow = kBase + threadIdx.y;
        if (brow < N) {
            sB[threadIdx.y][threadIdx.x*2    ] = B[brow*K + b0_off];
            sB[threadIdx.y][threadIdx.x*2 + 1] = B[brow*K + b1_off];
        } else {
            sB[threadIdx.y][threadIdx.x*2    ] = 0;
            sB[threadIdx.y][threadIdx.x*2 + 1] = 0;
        }

        // 4c) Sync *all* threads before using shared memory
        __syncthreads();

        // 4d) Compute partial sums over this tile
        int limit = min(TILE, N - kBase);
        for (int k = 0; k < limit; ++k) {
            int A0 = sA[threadIdx.y*2    ][k];
            int A1 = sA[threadIdx.y*2 + 1][k];
            int B0 = sB[k][threadIdx.x*2    ];
            int B1 = sB[k][threadIdx.x*2 + 1];
            sum += A0*B0 + A0*B1 + A1*B0 + A1*B1;
        }

        // 4e) Sync again before the next load
        __syncthreads();
    }

    // 5) Write the final result
    C[out_row * (K/2) + out_col] = sum;
}

/* GPU Optimized Function */
void rmm_gpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    /* Cuda events for calculating elapsed time */
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);

    // Allocate device memory
    int *d_matA, *d_matB, *d_matC;
    hipMalloc(&d_matA, M * N * sizeof(int));
    hipMalloc(&d_matB, N * K * sizeof(int));
    hipMalloc(&d_matC, (M/2) * (K/2) * sizeof(int));

    hipEventRecord(cpy_H2D_start);
    // Copy input matrices to device
    hipMemcpy(d_matA, matA, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matB, matB, N * K * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    // Calculate grid and block dimensions
    dim3 blockDim(TILE, TILE);  // TILExTILE threads per block
    dim3 gridDim((K/2 + TILE - 1) / TILE, 
                 (M/2 + TILE - 1) / TILE);

    hipEventRecord(comp_start);
    // Launch kernel
    rmm_kernel<<<gridDim, blockDim>>>(d_matA, d_matB, d_matC, M, N, K);
    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    // Copy result back to host
    hipMemcpy(matC, d_matC, (M/2) * (K/2) * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    /* Display timing statistics */
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "RMM operation takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;
}