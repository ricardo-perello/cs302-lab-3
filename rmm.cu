/*
============================================================================
Filename    : rmm.cu
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

/* CPU Baseline */
void rmm_cpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    for(int idx = 0; idx < M/2; idx++) {
        for(int jdx = 0; jdx < K/2; jdx++) {
            matC[idx*(K/2) + jdx] = 0;
            for(int aoff = 0; aoff < 2; aoff++) {
                for(int boff = 0; boff < 2; boff++) {
                    for(int kdx = 0; kdx < N; kdx++) {
                        matC[idx*(K/2) + jdx] += matA[(idx*2 + aoff)*N + kdx] * matB[kdx*K + jdx*2 + boff];
                    }
                }
            }
        }
    }
}

#define TILE 16

__global__ void rmm_kernel(
    const int* __restrict__ A,
    const int* __restrict__ B,
          int* __restrict__ C,
    int M, int N, int K)
{
    // global-reduced-matrix coords
    int out_row = blockIdx.y * TILE + threadIdx.y;  // 0 .. M/2-1
    int out_col = blockIdx.x * TILE + threadIdx.x;  // 0 .. K/2-1

    // mark who's actually computing a valid C element
    bool active = (out_row < M/2 && out_col < K/2);

    // precompute the two A-row bases and two B-col offsets (even if inactive)
    int a0_base = (out_row*2    ) * N;
    int a1_base = (out_row*2 + 1) * N;
    int b0_off  = out_col*2;
    int b1_off  = b0_off + 1;

    int sum = 0;

    // shared-memory tiles: 2 rows × TILE columns, and TILE rows × 2 columns
    __shared__ int sA[2*TILE][TILE];
    __shared__ int sB[TILE][2*TILE];

    int numTiles = (N + TILE - 1) / TILE;
    for (int t = 0; t < numTiles; ++t) {
        int kBase = t * TILE;

        // --- load A (two rows) ---
        int col = kBase + threadIdx.x;
        if (col < N) {
            sA[threadIdx.y*2    ][threadIdx.x] = A[a0_base + col];
            sA[threadIdx.y*2 + 1][threadIdx.x] = A[a1_base + col];
        } else {
            sA[threadIdx.y*2    ][threadIdx.x] = 0;
            sA[threadIdx.y*2 + 1][threadIdx.x] = 0;
        }

        // --- load B (two cols) ---
        int brow = kBase + threadIdx.y;
        if (brow < N) {
            sB[threadIdx.y][threadIdx.x*2    ] = B[brow*K + b0_off];
            sB[threadIdx.y][threadIdx.x*2 + 1] = B[brow*K + b1_off];
        } else {
            sB[threadIdx.y][threadIdx.x*2    ] = 0;
            sB[threadIdx.y][threadIdx.x*2 + 1] = 0;
        }

        // *all* threads sync before we read from sA/sB
        __syncthreads();

        // accumulate only if this thread is active
        if (active) {
            int limit = min(TILE, N - kBase);
            // Unroll the inner loop by a factor of 4
            #pragma unroll 4
            for (int k = 0; k < limit; k += 4) {
                // Process 4 elements at a time
                if (k + 0 < limit) {
                    int A0_0 = sA[threadIdx.y*2    ][k + 0];
                    int A1_0 = sA[threadIdx.y*2 + 1][k + 0];
                    int B0_0 = sB[k + 0][threadIdx.x*2    ];
                    int B1_0 = sB[k + 0][threadIdx.x*2 + 1];
                    sum += A0_0*B0_0 + A0_0*B1_0 + A1_0*B0_0 + A1_0*B1_0;
                }
                if (k + 1 < limit) {
                    int A0_1 = sA[threadIdx.y*2    ][k + 1];
                    int A1_1 = sA[threadIdx.y*2 + 1][k + 1];
                    int B0_1 = sB[k + 1][threadIdx.x*2    ];
                    int B1_1 = sB[k + 1][threadIdx.x*2 + 1];
                    sum += A0_1*B0_1 + A0_1*B1_1 + A1_1*B0_1 + A1_1*B1_1;
                }
                if (k + 2 < limit) {
                    int A0_2 = sA[threadIdx.y*2    ][k + 2];
                    int A1_2 = sA[threadIdx.y*2 + 1][k + 2];
                    int B0_2 = sB[k + 2][threadIdx.x*2    ];
                    int B1_2 = sB[k + 2][threadIdx.x*2 + 1];
                    sum += A0_2*B0_2 + A0_2*B1_2 + A1_2*B0_2 + A1_2*B1_2;
                }
                if (k + 3 < limit) {
                    int A0_3 = sA[threadIdx.y*2    ][k + 3];
                    int A1_3 = sA[threadIdx.y*2 + 1][k + 3];
                    int B0_3 = sB[k + 3][threadIdx.x*2    ];
                    int B1_3 = sB[k + 3][threadIdx.x*2 + 1];
                    sum += A0_3*B0_3 + A0_3*B1_3 + A1_3*B0_3 + A1_3*B1_3;
                }
            }
        }

        // *all* threads sync before we overwrite sA/sB
        __syncthreads();
    }

    // finally write C if active using atomic add
    if (active) {
        atomicAdd(&C[out_row * (K/2) + out_col], sum);
    }
}

/* GPU Optimized Function */
void rmm_gpu(int *matA, int *matB, int *matC, int M, int N, int K)
{
    /* Cuda events for calculating elapsed time */
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);

    // Create two CUDA streams for overlapping operations
    hipStream_t s1, s2;
    hipStreamCreate(&s1);
    hipStreamCreate(&s2);

    // Allocate pinned host memory
    int *h_matA, *h_matB, *h_matC;
    hipHostMalloc(&h_matA, M * N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_matB, N * K * sizeof(int), hipHostMallocDefault);
    hipHostMalloc(&h_matC, (M/2) * (K/2) * sizeof(int), hipHostMallocDefault);

    // Copy input data to pinned memory
    memcpy(h_matA, matA, M * N * sizeof(int));
    memcpy(h_matB, matB, N * K * sizeof(int));

    // Allocate device memory
    int *d_matA, *d_matB, *d_matC;
    hipMalloc(&d_matA, M * N * sizeof(int));
    hipMalloc(&d_matB, N * K * sizeof(int));
    hipMalloc(&d_matC, (M/2) * (K/2) * sizeof(int));

    // Initialize output matrix to zero
    hipMemset(d_matC, 0, (M/2) * (K/2) * sizeof(int));

    // Calculate grid and block dimensions
    dim3 blockDim(TILE, TILE);
    dim3 gridDim((K/2 + TILE - 1) / TILE, 
                 (M/2 + TILE - 1) / TILE);

    // Split the computation into two halves along the N dimension
    int N_half = N / 2;
    int N_remainder = N % 2;

    hipEventRecord(cpy_H2D_start);
    
    // Stream 1: First half of the computation
    hipMemcpyAsync(d_matA, h_matA, M * N_half * sizeof(int), hipMemcpyHostToDevice, s1);
    hipMemcpyAsync(d_matB, h_matB, N_half * K * sizeof(int), hipMemcpyHostToDevice, s1);
    
    // Stream 2: Second half of the computation
    hipMemcpyAsync(d_matA + M * N_half, h_matA + M * N_half, 
                    M * (N_half + N_remainder) * sizeof(int), hipMemcpyHostToDevice, s2);
    hipMemcpyAsync(d_matB + N_half * K, h_matB + N_half * K, 
                    (N_half + N_remainder) * K * sizeof(int), hipMemcpyHostToDevice, s2);

    hipEventRecord(comp_start);

    // Launch kernels on both streams
    rmm_kernel<<<gridDim, blockDim, 0, s1>>>(d_matA, d_matB, d_matC, M, N_half, K);
    rmm_kernel<<<gridDim, blockDim, 0, s2>>>(d_matA + M * N_half, d_matB + N_half * K, 
                                            d_matC, M, N_half + N_remainder, K);

    hipEventRecord(comp_end);

    // Copy results back to host
    hipEventRecord(cpy_D2H_start);
    hipMemcpyAsync(h_matC, d_matC, (M/2) * (K/2) * sizeof(int), hipMemcpyDeviceToHost, s1);
    hipEventRecord(cpy_D2H_end);

    // Synchronize all operations
    hipStreamSynchronize(s1);
    hipStreamSynchronize(s2);

    // Copy final result back to original memory
    memcpy(matC, h_matC, (M/2) * (K/2) * sizeof(int));

    // Free device memory
    hipFree(d_matA);
    hipFree(d_matB);
    hipFree(d_matC);

    // Free pinned host memory
    hipHostFree(h_matA);
    hipHostFree(h_matB);
    hipHostFree(h_matC);

    // Destroy streams
    hipStreamDestroy(s1);
    hipStreamDestroy(s2);

    /* Display timing statistics */
    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout << "Host to Device MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout << "RMM operation takes " << setprecision(4) << time/1000 << "s" << endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout << "Device to Host MemCpy takes " << setprecision(4) << time/1000 << "s" << endl;
}